#include "hip/hip_runtime.h"
/*
 * ENVISAT and ERS ASAR instrument focusser for QA4EO activity (c)
 * by CGI Estonia AS
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */



#include "device_padded_image.cuh"


/**
 * This program is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License as published by the Free
 * Software Foundation; either version 3 of the License, or (at your option)
 * any later version.
 * This program is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
 * more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, see http://www.gnu.org/licenses/
 */


#include <numeric>
#include <vector>

#include <hip/hip_complex.h>

#include "cuda_cleanup.h"

namespace {
    __global__ void ZeroFillPaddingsKernel(hipfftComplex* data, int x_size, int y_size, int x_stride, int y_stride) {
        const int x = threadIdx.x + blockIdx.x * blockDim.x;
        const int y = threadIdx.y + blockIdx.y * blockDim.y;

        const int idx = x + y * x_stride;

        if ((x >= x_size && x < x_stride) && (y >= y_size && y < y_stride)) {
            data[idx] = {};
        }
    }

    __global__ void MultiplyDataKernel(hipfftComplex* data, int x_size, int y_size, int x_stride, float multiplier) {
        const int x = threadIdx.x + blockIdx.x * blockDim.x;
        const int y = threadIdx.y + blockIdx.y * blockDim.y;

        const int idx = x + y * x_stride;

        if (x < x_size && y < y_size) {
            data[idx].x *= multiplier;
            data[idx].y *= multiplier;
        }
    }

    __global__ void NaNZeroKernel(hipfftComplex* data, int x_size, int y_size) {
        const int x = threadIdx.x + blockIdx.x * blockDim.x;
        const int y = threadIdx.y + blockIdx.y * blockDim.y;

        const int idx = x + y * x_size;

        if (x < x_size && y < y_size) {
            if(std::isnan(data[idx].x))
            {
                data[idx].x = 0.0f;
            }
            if(std::isnan(data[idx].y))
            {
                data[idx].y = 0.0f;
            }
        }
    }

// original code from with modification to use hipfftComplex instead
// https://github.com/JonathanWatkins/CUDA/blob/master/NvidiaCourse/Exercises/transpose/transpose.cu
    constexpr int TRANSPOSE_BLOCK_DIM = 16;
    __global__ void TransposeKernel(hipfftComplex* idata, hipfftComplex* odata, int width, int height) {
        __shared__ hipfftComplex block[TRANSPOSE_BLOCK_DIM][TRANSPOSE_BLOCK_DIM + 1];

        // read the matrix tile into shared memory
        // load one element per thread from device memory (idata) and store it
        // in transposed order in block[][]
        int xIndex = blockIdx.x * TRANSPOSE_BLOCK_DIM + threadIdx.x;
        int yIndex = blockIdx.y * TRANSPOSE_BLOCK_DIM + threadIdx.y;
        if ((xIndex < width) && (yIndex < height)) {
            unsigned int index_in = yIndex * width + xIndex;
            block[threadIdx.y][threadIdx.x] = idata[index_in];
        }

        // synchronise to ensure all writes to block[][] have completed
        __syncthreads();

        // write the transposed matrix tile to global memory (odata) in linear order
        xIndex = blockIdx.y * TRANSPOSE_BLOCK_DIM + threadIdx.x;
        yIndex = blockIdx.x * TRANSPOSE_BLOCK_DIM + threadIdx.y;
        if ((xIndex < height) && (yIndex < width)) {
            int index_out = yIndex * height + xIndex;
            odata[index_out] = block[threadIdx.x][threadIdx.y];
        }
    }

    constexpr int REDUCE_BLOCK_SIZE = 1024;

    __global__ void ReduceIntensity(const hipfftComplex* data, int x_size, int y_size, int x_stride, float* result) {
        __shared__ float shared_acc[REDUCE_BLOCK_SIZE];
        const int y_start = blockIdx.y;
        const int x_start = threadIdx.x;
        const int shared_idx = x_start;
        const int result_idx = blockIdx.y;
        const int y_step = gridDim.y;
        const int x_step = blockDim.x;

        double acc = {};
        for (int y = y_start; y < y_size; y += y_step) {
            float inner_acc = {};
            for (int x = x_start; x < x_size; x += x_step) {
                int idx = (y * x_stride) + x;
                auto sample = data[idx];
                const float i = sample.x;
                const float q = sample.y;
                inner_acc += i * i + q * q;
            }
            acc += inner_acc;
        }

        shared_acc[shared_idx] = acc;
        __syncthreads();

        for (int s = REDUCE_BLOCK_SIZE / 2; s > 0; s /= 2) {
            if (shared_idx < s) {
                float t = shared_acc[shared_idx] + shared_acc[shared_idx + s];
                shared_acc[shared_idx] = t;
            }
            __syncthreads();
        }

        if (shared_idx == 0) {
            result[result_idx] = shared_acc[0];
        }
    }


    __global__ void ReduceStdDevDiffSquares(const hipfftComplex* src, int x_size, int y_size, int x_stride, float i_mean, float q_mean, float* i_result_arr,
                                    float* q_result_arr) {
        __shared__ float i_shared_acc[REDUCE_BLOCK_SIZE];
        __shared__ float q_shared_acc[REDUCE_BLOCK_SIZE];

        const int shared_idx = threadIdx.x;
        const int result_idx = blockIdx.y;
        const int y_start = blockIdx.y;
        const int x_start = threadIdx.x;
        const int y_step = gridDim.y;
        const int x_step = blockDim.x;

        double i_diff_sq_sum = 0.0;
        double q_diff_sq_sum = 0.0;
        for (int y = y_start; y < y_size; y += y_step) {
            for (int x = x_start; x < x_size; x += x_step) {
                int idx = (y * x_stride) + x;
                auto sample = src[idx];
                float i = sample.x;
                float q = sample.y;

                float diff_i = i - i_mean;
                float diff_q = q - q_mean;

                if (!std::isnan(i) && !std::isnan(q)) {
                    i_diff_sq_sum += diff_i * diff_i;
                    q_diff_sq_sum += diff_q * diff_q;
                }
            }
        }

        // store each thread's result in shared memory, ensure all threads reach this point
        i_shared_acc[shared_idx] = i_diff_sq_sum;
        q_shared_acc[shared_idx] = q_diff_sq_sum;
        __syncthreads();

        // reduce the shared memory array to the first element
        // Reduction #3: Sequential Addressing from NVidia Optimizing Parallel Reduction
        for (int s = REDUCE_BLOCK_SIZE / 2; s > 0; s /= 2) {
            if (shared_idx < s) {
                i_shared_acc[shared_idx] += i_shared_acc[shared_idx + s];
                q_shared_acc[shared_idx] += q_shared_acc[shared_idx + s];
            }
            __syncthreads();
        }

        // write the final result to global memory
        if (shared_idx == 0) {
            i_result_arr[result_idx] = i_shared_acc[0];
            q_result_arr[result_idx] = q_shared_acc[0];
        }
    }

    __global__ void ReduceMean(const hipfftComplex* src, int x_size, int y_size, int x_stride, float* i_result_arr,
                                            float* q_result_arr) {
        __shared__ float i_shared_acc[REDUCE_BLOCK_SIZE];
        __shared__ float q_shared_acc[REDUCE_BLOCK_SIZE];

        const int shared_idx = threadIdx.x;
        const int result_idx = blockIdx.y;
        const int y_start = blockIdx.y;
        const int x_start = threadIdx.x;
        const int y_step = gridDim.y;
        const int x_step = blockDim.x;

        double i_sum = 0.0;
        double q_sum = 0.0;
        for (int y = y_start; y < y_size; y += y_step) {
            for (int x = x_start; x < x_size; x += x_step) {
                int idx = (y * x_stride) + x;
                auto sample = src[idx];
                float i = sample.x;
                float q = sample.y;

                if (!std::isnan(i) && !std::isnan(q)) {
                    i_sum += i;
                    q_sum += q;
                }
            }
        }

        // store each thread's result in shared memory, ensure all threads reach this point
        i_shared_acc[shared_idx] = i_sum;
        q_shared_acc[shared_idx] = q_sum;
        __syncthreads();

        // reduce the shared memory array to the first element
        // Reduction #3: Sequential Addressing from NVidia Optimizing Parallel Reduction
        for (int s = REDUCE_BLOCK_SIZE / 2; s > 0; s /= 2) {
            if (shared_idx < s) {
                i_shared_acc[shared_idx] += i_shared_acc[shared_idx + s];
                q_shared_acc[shared_idx] += q_shared_acc[shared_idx + s];
            }
            __syncthreads();
        }

        // write the final result to global memory
        if (shared_idx == 0) {
            i_result_arr[result_idx] = i_shared_acc[0];
            q_result_arr[result_idx] = q_shared_acc[0];
        }
    }


}  // namespace


    void DevicePaddedImage::MultiplyData(float multiplier) {
        dim3 block_size(16, 16);
        dim3 grid_size((x_stride_ + 15) / 16, (y_stride_ + 15) / 16);
        MultiplyDataKernel<<<grid_size, block_size>>>(d_data_, x_size_, y_size_, x_stride_, multiplier);
    }
    void DevicePaddedImage::ZeroFillPaddings() {
        dim3 block_size(16, 16);
        dim3 grid_size((x_stride_ + 15) / 16, (y_stride_ + 15) / 16);
        ZeroFillPaddingsKernel<<<grid_size, block_size>>>(d_data_, x_size_, y_size_, x_stride_, y_stride_);
    }

    void DevicePaddedImage::Transpose() {
        // TODO(priit) investigate inplace rectangular transpose?
        hipfftComplex* d_transposed;
        CHECK_CUDA_ERR(hipMalloc(&d_transposed, TotalByteSize()));

        dim3 block_size(TRANSPOSE_BLOCK_DIM, TRANSPOSE_BLOCK_DIM);
        dim3 grid_size((x_stride_ + TRANSPOSE_BLOCK_DIM - 1) / TRANSPOSE_BLOCK_DIM,
                       (y_stride_ + TRANSPOSE_BLOCK_DIM - 1) / TRANSPOSE_BLOCK_DIM);

        TransposeKernel<<<grid_size, block_size>>>(d_data_, d_transposed, x_stride_, y_stride_);
        hipFree(d_data_);
        std::swap(x_stride_, y_stride_);
        std::swap(x_size_, y_size_);
        d_data_ = d_transposed;
    }

    void DevicePaddedImage::Transpose(CudaWorkspace& d_workspace) {
        VerifyExtMemorySize(TotalByteSize(), d_workspace.ByteSize());

        dim3 block_size(TRANSPOSE_BLOCK_DIM, TRANSPOSE_BLOCK_DIM);
        dim3 grid_size((x_stride_ + TRANSPOSE_BLOCK_DIM - 1) / TRANSPOSE_BLOCK_DIM,
                       (y_stride_ + TRANSPOSE_BLOCK_DIM - 1) / TRANSPOSE_BLOCK_DIM);

        TransposeKernel<<<grid_size, block_size>>>(d_data_, d_workspace.GetAs<hipfftComplex>(), x_stride_, y_stride_);
        std::swap(x_stride_, y_stride_);
        std::swap(x_size_, y_size_);

        // swap pointers, so transposed image is owned by this object, and the old memory is now usable by others
        auto* d_workspace_ptr = static_cast<hipfftComplex*>(d_workspace.ReleaseMemory());
        d_workspace.Reset(d_data_, TotalByteSize());
        std::swap(d_workspace_ptr, d_data_);
    }

    double DevicePaddedImage::CalcTotalIntensity(size_t sm_count) {
        size_t n_blocks = sm_count;
        dim3 grid_dim(1, n_blocks);
        float* d_block_sums;
        size_t byte_size = n_blocks * sizeof(float);
        CHECK_CUDA_ERR(hipMalloc(&d_block_sums, byte_size));
        CudaMallocCleanup  clean(d_block_sums);

        ReduceIntensity<<<grid_dim, REDUCE_BLOCK_SIZE>>>(d_data_, x_size_, y_size_, x_stride_, d_block_sums);
        std::vector<float> h_block_sums(n_blocks);
        CHECK_CUDA_ERR(hipMemcpy(h_block_sums.data(), d_block_sums, byte_size, hipMemcpyDeviceToHost));

        return std::accumulate(h_block_sums.begin(), h_block_sums.end(), 0.0);
    }

    void DevicePaddedImage::ZeroNaNs()
    {
        dim3 block_sz(16, 16);
        dim3 grid_sz((x_stride_ + 15) / 16, (y_stride_ + 15) / 16);

        NaNZeroKernel<<<grid_sz, block_sz>>>(d_data_, x_stride_, y_stride_);

    }

    hipfftComplex DevicePaddedImage::CalcStdDev(float i_mean, float q_mean, size_t total_samples) const
    {
        size_t n_blocks = 50;
        size_t byte_size = n_blocks * sizeof(float);


        dim3 grid_dim(1, n_blocks);

        float* i_diff_sum;
        float* q_diff_sum;
        CHECK_CUDA_ERR(hipMalloc(&i_diff_sum, byte_size));
        CudaMallocCleanup clean_i(i_diff_sum);

        CHECK_CUDA_ERR(hipMalloc(&q_diff_sum, byte_size));
        CudaMallocCleanup clean_q(q_diff_sum);

        ReduceStdDevDiffSquares<<<grid_dim, REDUCE_BLOCK_SIZE>>>(d_data_, x_size_, y_size_, x_stride_, i_mean, q_mean, i_diff_sum, q_diff_sum);

        // each thread block has summed it's part of squared mean differences


        // do the second part of reduction on cpu, beacuse this is not a speed bottleneck in the project
        std::vector<float> h_i_sum(n_blocks);
        CHECK_CUDA_ERR(hipMemcpy(h_i_sum.data(), i_diff_sum, byte_size, hipMemcpyDeviceToHost));
        std::vector<float> h_q_sum(n_blocks);
        CHECK_CUDA_ERR(hipMemcpy(h_q_sum.data(), q_diff_sum, byte_size, hipMemcpyDeviceToHost));

        hipfftComplex r = {};
        r.x = std::accumulate(h_i_sum.begin(), h_i_sum.end(), 0.0);
        r.y = std::accumulate(h_q_sum.begin(), h_q_sum.end(), 0.0);

        if (total_samples == 0) {
            total_samples = x_size_ * y_size_;
        }
        // variance
        r.x /= total_samples;
        r.y /= total_samples;

        // standard deviation
        r.x = sqrt(r.x);
        r.y = sqrt(r.y);

        return r;
    }

    hipfftComplex DevicePaddedImage::CalcMean(size_t total_samples) const
    {
        size_t n_blocks = 50;
        size_t byte_size = n_blocks * sizeof(float);

        dim3 grid_dim(1, n_blocks);

        float* i_sum;
        float* q_sum;
        CHECK_CUDA_ERR(hipMalloc(&i_sum, byte_size));
        CudaMallocCleanup clean_i(i_sum);

        CHECK_CUDA_ERR(hipMalloc(&q_sum, byte_size));
        CudaMallocCleanup clean_q(q_sum);

        ReduceMean<<<grid_dim, REDUCE_BLOCK_SIZE>>>(d_data_, x_size_, y_size_, x_stride_, i_sum, q_sum);

        // each thread block has summed it's part of I & Q channels


        // do the second part of reduction on cpu, beacuse this is not a speed bottleneck in the project
        std::vector<float> h_i_sum(n_blocks);
        CHECK_CUDA_ERR(hipMemcpy(h_i_sum.data(), i_sum, byte_size, hipMemcpyDeviceToHost));
        std::vector<float> h_q_sum(n_blocks);
        CHECK_CUDA_ERR(hipMemcpy(h_q_sum.data(), q_sum, byte_size, hipMemcpyDeviceToHost));

        hipfftComplex r = {};
        r.x = std::accumulate(h_i_sum.begin(), h_i_sum.end(), 0.0);
        r.y = std::accumulate(h_q_sum.begin(), h_q_sum.end(), 0.0);

        if (total_samples == 0) {
            total_samples = x_size_ * y_size_;
        }

        r.x /= total_samples;
        r.y /= total_samples;

        return r;
    }