#include "hip/hip_runtime.h"


#include "range_compression.cuh"

#include "cuda_util/cuda_cleanup.h"
#include "cuda_util/cuda_util.h"
#include "cuda_util/cufft_plan.h"
#include "util/checks.h"
// #include "math_utils.h"

__global__ void FrequencyDomainMultiply(hipfftComplex* data_fft, const hipfftComplex* chirp_fft, int range_fft_size,
                                        int azimuth_size) {
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    const int data_idx = y * range_fft_size + x;

    if (x < range_fft_size && y < azimuth_size) {
        hipfftComplex chirp_bin = chirp_fft[x];

        // Conjugate chirp FFT bin before multiplication
        chirp_bin = hipConjf(chirp_bin);

        hipfftComplex data_bin = data_fft[data_idx];

        data_fft[data_idx] = hipCmulf(data_bin, chirp_bin);
    }
}

__global__ void FinishRangeCompression(hipfftComplex* data_fft, int range_size, int cutoff, int azimuth_size,
                                       float multiplier) {
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    const int data_idx = y * range_size + x;

    if (x < cutoff && y < azimuth_size) {
        data_fft[data_idx].x *= multiplier;
        data_fft[data_idx].y *= multiplier;
    } else if (x < range_size && y < azimuth_size) {
        data_fft[data_idx] = {};
    }
}

void RangeCompression(DevicePaddedImage& data, const std::vector<std::complex<float>>& chirp_data, int chirp_samples,
                      CudaWorkspace& d_workspace) {
    const int range_fft_size = data.XStride();
    const int azimuth_size = data.YSize();
    const int chirp_size = static_cast<int>(chirp_data.size());

    if (range_fft_size != chirp_size) {
        throw std::logic_error("Range padding and chirp size mismatch");
    }

    // chirp host -> device
    hipfftComplex* d_chirp;
    static_assert(sizeof(hipfftComplex) == sizeof(chirp_data[0]));

    const size_t chirp_bsize = 8 * chirp_data.size();
    CHECK_CUDA_ERR(hipMalloc(&d_chirp, chirp_bsize));
    CudaMallocCleanup chirp_cleanup(d_chirp);
    CHECK_CUDA_ERR(hipMemcpy(d_chirp, chirp_data.data(), chirp_bsize, hipMemcpyHostToDevice));

    {
        // inplace FFT of chirp
        // NB! conjugation applied during frequency domain multiplication
        // TODO(priit) do this on CPU?
        hipfftHandle chirp_plan;
        CHECK_CUFFT_ERR(hipfftPlan1d(&chirp_plan, range_fft_size, HIPFFT_C2C, 1));
        CufftPlanCleanup plan_cleanup(chirp_plan);
        CHECK_CUFFT_ERR(hipfftExecC2C(chirp_plan, d_chirp, d_chirp, HIPFFT_FORWARD));
    }

    hipComplex* d_data = data.Data();
    // inplace FFT on each range row of the SAR data
    hipfftHandle range_fft_plan = PlanRangeFFT(range_fft_size, azimuth_size, false);
    CufftPlanCleanup range_fft_cleanup(range_fft_plan);
    CheckCufftSize(d_workspace.ByteSize(), range_fft_plan);
    CHECK_CUFFT_ERR(hipfftSetWorkArea(range_fft_plan, d_workspace.Get()));
    CHECK_CUFFT_ERR(hipfftExecC2C(range_fft_plan, d_data, d_data, HIPFFT_FORWARD));

    // matched filter via frequency domain multiplication
    dim3 block_size(16, 16);
    dim3 grid_size((range_fft_size + 15) / 16, (azimuth_size + 15) / 16);
    FrequencyDomainMultiply<<<grid_size, block_size>>>(d_data, d_chirp, range_fft_size, azimuth_size);

    // Back to time domain
    CHECK_CUFFT_ERR(hipfftExecC2C(range_fft_plan, d_data, d_data, HIPFFT_BACKWARD));

    const int new_range_size = data.XSize() - chirp_samples;
    data.SetXSize(new_range_size);

    // scale FFT -> IFFT roundtrip and zero memory of the padding
    FinishRangeCompression<<<grid_size, block_size>>>(d_data, range_fft_size, new_range_size, azimuth_size,
                                                      1.0 / range_fft_size);

    CHECK_CUDA_ERR(hipDeviceSynchronize());  // not needed at this point, but helps with debugging
    CHECK_CUDA_ERR(hipGetLastError());
}

namespace {
struct SRCArgs {
    // float Kr;
    float Vr;
    float carrier_frequency;
    float rsr;
    float prf;
    double R;
    float lambda;
};

__global__ void SRCKernel(hipfftComplex* data_2dfft, int range_size, int azimuth_size, SRCArgs args) {
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    const int data_idx = y * range_size + x;

    float range_bin_step = args.rsr / range_size;
    float azimuth_bin_step = args.prf / azimuth_size;

    float fr = 0.0f;
    if (y < (range_size / 2)) {
        fr = y * range_bin_step;

    } else {
        fr = (y - range_size) * range_bin_step;
    }

    float fn = 0.0f;
    if (y < (azimuth_size / 2)) {
        fn = y * azimuth_bin_step;

    } else {
        fn = (y - azimuth_size) * azimuth_bin_step;
    }

    float term = (args.lambda * args.lambda * fn * fn) / (4 * args.Vr * args.Vr);
    float D = sqrtf(1 - term);
    float f0 = args.carrier_frequency;
    double Ksrc = 2 * args.Vr * args.Vr * f0 * f0 * f0 * D * D * D;
    Ksrc /= (SOL * args.R * fn * fn);

    double phase = -M_PI * fr * fr / Ksrc;

    hipfftComplex val;
    float sin_val;
    float cos_val;
    sincos(phase, &sin_val, &cos_val);
    val.x = cos_val;
    val.y = sin_val;

    data_2dfft[data_idx] = hipCmulf(data_2dfft[data_idx], val);
}

}  // namespace

void SecondaryRangeCompression(DevicePaddedImage& img, const SARMetadata& sar_meta, CudaWorkspace& d_workspace) {
    hipfftComplex* d_data = img.Data();
    int rg_size = img.XStride();
    int az_size = img.YStride();
    hipfftHandle fft2d = Plan2DFFT(rg_size, az_size);

    CufftPlanCleanup fft_cleanup(fft2d);

    CHECK_CUFFT_ERR(hipfftSetWorkArea(fft2d, d_workspace.Get()));
    CHECK_CUFFT_ERR(hipfftExecC2C(fft2d, d_data, d_data, HIPFFT_FORWARD));

    dim3 block_size(16, 16);
    dim3 grid_size((rg_size + 15) / 16, (az_size + 15) / 16);
    SRCArgs args = {};

    args.Vr = CalcVr(sar_meta, rg_size / 2);
    args.lambda = sar_meta.wavelength;
    args.carrier_frequency = sar_meta.carrier_frequency;
    args.prf = sar_meta.pulse_repetition_frequency;
    args.R = CalcR0(sar_meta, rg_size / 2);
    args.rsr = sar_meta.chirp.range_sampling_rate;
    SRCKernel<<<grid_size, block_size>>>(d_data, rg_size, az_size, args);

    CHECK_CUFFT_ERR(hipfftExecC2C(fft2d, d_data, d_data, HIPFFT_BACKWARD));

    //img.MultiplyData(1.0f/(rg_size * az_size));

    hipDeviceSynchronize();
}