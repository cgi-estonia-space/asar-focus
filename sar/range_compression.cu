#include "hip/hip_runtime.h"
/**
* ENVISAT and ERS ASAR instrument focusser for QA4EO activity (c) by CGI Estonia AS
*
* ENVISAT and ERS ASAR instrument focusser for QA4EO activity is licensed under a
* Creative Commons Attribution-ShareAlike 4.0 International License.
*
* You should have received a copy of the license along with this
* work. If not, see http://creativecommons.org/licenses/by-sa/4.0/
 */

#include "range_compression.cuh"

#include "cuda_util/cuda_cleanup.h"
#include "cuda_util/cuda_util.h"
#include "cuda_util/cufft_plan.h"
#include "checks.h"

__global__ void FrequencyDomainMultiply(hipfftComplex* data_fft, const hipfftComplex* chirp_fft, int range_fft_size,
                                        int azimuth_size) {
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    const int data_idx = y * range_fft_size + x;

    if (x < range_fft_size && y < azimuth_size) {
        hipfftComplex chirp_bin = chirp_fft[x];

        // Conjugate chirp FFT bin before multiplication
        chirp_bin = hipConjf(chirp_bin);

        hipfftComplex data_bin = data_fft[data_idx];

        data_fft[data_idx] = hipCmulf(data_bin, chirp_bin);
    }
}

__global__ void FinishRangeCompression(hipfftComplex* data_fft, int range_size, int cutoff, int azimuth_size,
                                       float multiplier) {
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    const int data_idx = y * range_size + x;

    if (x < cutoff && y < azimuth_size) {
        data_fft[data_idx].x *= multiplier;
        data_fft[data_idx].y *= multiplier;
    } else if (x < range_size && y < azimuth_size) {
        data_fft[data_idx] = {};
    }
}

void RangeCompression(DevicePaddedImage& data, const std::vector<std::complex<float>>& chirp_data, int chirp_samples,
                      CudaWorkspace& d_workspace) {
    const int range_fft_size = data.XStride();
    const int azimuth_size = data.YSize();
    const int chirp_size = static_cast<int>(chirp_data.size());

    if (range_fft_size != chirp_size) {
        throw std::logic_error("Range padding and chirp size mismatch");
    }

    // chirp host -> device
    hipfftComplex* d_chirp;
    static_assert(sizeof(hipfftComplex) == sizeof(chirp_data[0]));

    const size_t chirp_bsize = 8 * chirp_data.size();
    CHECK_CUDA_ERR(hipMalloc(&d_chirp, chirp_bsize));
    CudaMallocCleanup chirp_cleanup(d_chirp);
    CHECK_CUDA_ERR(hipMemcpy(d_chirp, chirp_data.data(), chirp_bsize, hipMemcpyHostToDevice));

    {
        // inplace FFT of chirp
        // NB! conjugation applied during frequency domain multiplication
        // TODO(priit) do this on CPU?
        hipfftHandle chirp_plan;
        CHECK_CUFFT_ERR(hipfftPlan1d(&chirp_plan, range_fft_size, HIPFFT_C2C, 1));
        CufftPlanCleanup plan_cleanup(chirp_plan);
        CHECK_CUFFT_ERR(hipfftExecC2C(chirp_plan, d_chirp, d_chirp, HIPFFT_FORWARD));
    }

    hipComplex* d_data = data.Data();
    // inplace FFT on each range row of the SAR data
    hipfftHandle range_fft_plan = PlanRangeFFT(range_fft_size, azimuth_size, false);
    CufftPlanCleanup range_fft_cleanup(range_fft_plan);
    CheckCufftSize(d_workspace.ByteSize(), range_fft_plan);
    CHECK_CUFFT_ERR(hipfftSetWorkArea(range_fft_plan, d_workspace.Get()));
    CHECK_CUFFT_ERR(hipfftExecC2C(range_fft_plan, d_data, d_data, HIPFFT_FORWARD));

    // matched filter via frequency domain multiplication
    dim3 block_size(16, 16);
    dim3 grid_size((range_fft_size + 15) / 16, (azimuth_size + 15) / 16);
    FrequencyDomainMultiply<<<grid_size, block_size>>>(d_data, d_chirp, range_fft_size, azimuth_size);

    // Back to time domain
    CHECK_CUFFT_ERR(hipfftExecC2C(range_fft_plan, d_data, d_data, HIPFFT_BACKWARD));

    const int new_range_size = data.XSize() - chirp_samples;
    data.SetXSize(new_range_size);

    // scale FFT -> IFFT roundtrip and zero memory of the padding
    FinishRangeCompression<<<grid_size, block_size>>>(d_data, range_fft_size, new_range_size, azimuth_size,
                                                      1.0 / range_fft_size);

    CHECK_CUDA_ERR(hipDeviceSynchronize());  // not needed at this point, but helps with debugging
    CHECK_CUDA_ERR(hipGetLastError());
}
